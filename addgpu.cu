
#include <hip/hip_runtime.h>
__global__ void add_gpu(int n, float* x, float* y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}


int main (void)
{
    int N = 1 << 20;
    float* x;
    float* y;
    
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    add_gpu<<<1,1>>>(N, x, y);

    hipFree(x);
    hipFree(y);
}